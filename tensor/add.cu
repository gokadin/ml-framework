#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "tensor.h"
#include "cudautils.h"

const int BLOCK_SIZE = 1024;

__global__ void add(float *a, float* b, float *target, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < size; i += stride) {
        target[i] = a[i] + b[i];
    }
}

extern "C" {

    __declspec(dllexport) int gpu_add_forward(TENSOR *target, TENSOR* a, TENSOR *b) {
        float* gpu_a;
        size_t a_size = a->mat_shape->size * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_a, a_size));
        checkCudaErr(hipMemcpy(gpu_a, &a->data[0], a_size, hipMemcpyHostToDevice));

        float* gpu_b;
        size_t b_size = b->mat_shape->size * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_b, b_size));
        checkCudaErr(hipMemcpy(gpu_b, &b->data[0], b_size, hipMemcpyHostToDevice));

        float* gpu_target;
        size_t target_size = target->mat_shape->size * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_target, target_size));

        dim3 blockSize = dim3(BLOCK_SIZE);
        dim3 gridSize = dim3((target->mat_shape->size + BLOCK_SIZE - 1) / BLOCK_SIZE);
        add<<<gridSize, blockSize>>>(gpu_a, gpu_b, gpu_target, target->mat_shape->size);
        checkCudaKernelErr("add", blockSize, gridSize);

        checkCudaErr(hipMemcpy(&target->data[0], gpu_target, target_size, hipMemcpyDeviceToHost));

        hipFree(gpu_a);
        hipFree(gpu_b);
        hipFree(gpu_target);

        return 0;
    }

}

