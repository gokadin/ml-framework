#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "tensor.h"

__global__ void add(float *a, float* b, float *target, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < size; i += stride) {
        target[i] = a[i] + b[i];
    }
}

extern "C" {

    void add(TENSOR *a, TENSOR* b, TENSOR *target) {
        int size = a->mat_shape.x * a->mat_shape.y;
        int msize = size * sizeof(float);
        float* gpu_a;
        float* gpu_b;
        float* gpu_target;

        hipMalloc((void**)&gpu_a, msize);
        hipMemcpy(gpu_a, &a->data[0], msize, hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_b, msize);
        hipMemcpy(gpu_b, &b->data[0], msize, hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_target, msize);

        dim3 blockSize = dim3(1024);
        dim3 gridSize = dim3((size + blockSize.x - 1) / blockSize.x);
        add<<<gridSize, blockSize>>>(gpu_a, gpu_b, gpu_target, size);

        hipMemcpy(&target->data[0], gpu_target, msize, hipMemcpyDeviceToHost);

        hipFree(gpu_a);
        hipFree(gpu_b);
        hipFree(gpu_target);
    }

}

