#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "tensor.h"
#include "cudautils.h"

const int BLOCK_SIZE = 1024;

__global__ void sum0(float *a, float *target, int width, int height)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < width)
    {
        for (int i = 0; i < height; i++)
        {
            sum += a[col + i * width];
        }
        target[col] = sum;
    }
}

__global__ void sum1(float *a, float *target, int width, int height)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (row < height)
    {
        for (int i = 0; i < width; i++)
        {
            sum += a[row * width + i];
        }
        target[row] = sum;
    }
}

extern "C" {

    __declspec(dllexport) void gpu_sum_forward(TENSOR *a, int axis, TENSOR *target) {
        float* gpu_a;
        size_t a_size = a->mat_size * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_a, a_size));
        checkCudaErr(hipMemcpy(gpu_a, &a->data[0], a_size, hipMemcpyHostToDevice));

        float* gpu_target;
        size_t size_target;

        if (axis == 0) {
            size_target = a->mat_shape[1] * sizeof(float);
            checkCudaErr(hipMalloc((void**)&gpu_target, size_target));

            dim3 blockSize(BLOCK_SIZE);
            dim3 gridSize((a->mat_shape[1] + BLOCK_SIZE - 1) / BLOCK_SIZE);
            sum0<<<gridSize, blockSize>>>(gpu_a, gpu_target, a->mat_shape[1], a->mat_shape[0]);
            checkCudaKernelErr("sum0", blockSize, gridSize);
        } else if (axis == 1) {
            size_target = a->mat_shape[0] * sizeof(float);
            checkCudaErr(hipMalloc((void**)&gpu_target, size_target));

            dim3 blockSize(BLOCK_SIZE);
            dim3 gridSize((a->mat_shape[0] + BLOCK_SIZE - 1) / BLOCK_SIZE);
            sum1<<<gridSize, blockSize>>>(gpu_a, gpu_target, a->mat_shape[1], a->mat_shape[0]);
            checkCudaKernelErr("sum1", blockSize, gridSize);
        }

        checkCudaErr(hipMemcpy(&target->data[0], gpu_target, size_target, hipMemcpyDeviceToHost));

        hipFree(gpu_a);
        hipFree(gpu_target);
    }

}

