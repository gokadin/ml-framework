#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "tensor.h"

const int BLOCK_SIZE = 32;

__global__ void linear(float *a, float *x, float *b, float *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * x[i * k + col];
        }
        c[row * k + col] = sum + b[col];
    }
}

extern "C" {

    void gpu_linear(const TENSOR *a, const TENSOR *x, const TENSOR* b, TENSOR *target)
    {
        float* gpu_a;
        size_t size = a->mat_shape->x * a->mat_shape->y * sizeof(float);
        hipMalloc((void**)&gpu_a, size);
        hipMemcpy(gpu_a, &a->data[0], size, hipMemcpyHostToDevice);

        float* gpu_b;
        size_t size_b = b->mat_shape->y * sizeof(float);
        hipMalloc((void**)&gpu_b, size_b);
        hipMemcpy(gpu_b, &b->data[0], size_b, hipMemcpyHostToDevice);

        float* gpu_x;
        size = x->mat_shape->x * x->mat_shape->y * sizeof(float);
        hipMalloc((void**)&gpu_x, size);
        hipMemcpy(gpu_x, &x->data[0], size, hipMemcpyHostToDevice);

        float* gpu_target;
        size = target->mat_shape->x * target->mat_shape->y * sizeof(float);
        hipMalloc(&gpu_target, size);

        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridSize((x->mat_shape->y + BLOCK_SIZE - 1) / BLOCK_SIZE, (a->mat_shape->x + BLOCK_SIZE - 1) / BLOCK_SIZE);
        linear<<<gridSize, blockSize>>>(gpu_a, gpu_x, gpu_b, gpu_target, a->mat_shape->x, a->mat_shape->y, x->mat_shape->y);

        hipMemcpy(&target->data[0], gpu_target, size, hipMemcpyDeviceToHost);

        hipFree(gpu_a);
        hipFree(gpu_x);
        hipFree(gpu_b);
        hipFree(gpu_target);
    }

}

