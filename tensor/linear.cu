#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cudautils.h"
#include "tensor.h"
#include "matmul.cuh"
#include "sum.cuh"

const int BLOCK_SIZE = 32;
const int BLOCK_SIZE_SUM = 1024;

__global__ void linear(float *a, float *x, float *b, float *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * x[i * k + col];
        }
        c[row * k + col] = sum + b[col];
    }
}

extern "C" {

    int gpu_linear_forward(const TENSOR *a, const TENSOR *x, const TENSOR* b, TENSOR *target)
    {
        float* gpu_a;
        size_t a_size = a->mat_shape->size * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_a, a_size));
        checkCudaErr(hipMemcpy(gpu_a, &a->data[0], a_size, hipMemcpyHostToDevice));

        float* gpu_b;
        size_t b_size = b->mat_shape->size * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_b, b_size));
        checkCudaErr(hipMemcpy(gpu_b, &b->data[0], b_size, hipMemcpyHostToDevice));

        float* gpu_x;
        size_t x_size = x->mat_shape->size * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_x, x_size));
        checkCudaErr(hipMemcpy(gpu_x, &x->data[0], x_size, hipMemcpyHostToDevice));

        float* gpu_target;
        size_t target_size = target->mat_shape->size * sizeof(float);
        checkCudaErr(hipMalloc(&gpu_target, target_size));

        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridSize((x->mat_shape->y + BLOCK_SIZE - 1) / BLOCK_SIZE, (a->mat_shape->x + BLOCK_SIZE - 1) / BLOCK_SIZE);
        linear<<<gridSize, blockSize>>>(gpu_a, gpu_x, gpu_b, gpu_target, a->mat_shape->x, a->mat_shape->y, x->mat_shape->y);
        checkCudaErr(hipPeekAtLastError());

        checkCudaErr(hipMemcpy(&target->data[0], gpu_target, target_size, hipMemcpyDeviceToHost));

        hipFree(gpu_a);
        hipFree(gpu_x);
        hipFree(gpu_b);
        hipFree(gpu_target);

        return 0;
    }

    int gpu_linear_backward(const TENSOR *tensor, const TENSOR *a, const TENSOR *x, TENSOR *b)
    {
        float* gpu_tensor_grad;
        size_t gpu_tensor_grad_size = tensor->grad_shape->x * tensor->grad_shape->y * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_tensor_grad, gpu_tensor_grad_size));
        checkCudaErr(hipMemcpy(gpu_tensor_grad, &tensor->grad[0], gpu_tensor_grad_size, hipMemcpyHostToDevice));

        float* gpu_a;
        size_t a_size = a->mat_shape->x * a->mat_shape->y * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_a, a_size));
        checkCudaErr(hipMemcpy(gpu_a, &a->data[0], a_size, hipMemcpyHostToDevice));

        float* gpu_a_grad;
        checkCudaErr(hipMalloc(&gpu_a_grad, a_size));

        float* gpu_b;
        size_t b_size = b->mat_shape->x * b->mat_shape->y * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_b, b_size));
        checkCudaErr(hipMemcpy(gpu_b, &b->data[0], b_size, hipMemcpyHostToDevice));

        float* gpu_b_grad;
        size_t b_grad_size = b->grad_shape->x * b->grad_shape->y * sizeof(float);
        checkCudaErr(hipMalloc(&gpu_b_grad, b_size));

        float* gpu_x;
        size_t x_size = x->mat_shape->x * x->mat_shape->y * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_x, x_size));
        checkCudaErr(hipMemcpy(gpu_x, &x->data[0], x_size, hipMemcpyHostToDevice));

        float* gpu_x_grad;
        checkCudaErr(hipMalloc(&gpu_x_grad, x_size));

        hipStream_t streamA, streamB;
        hipStreamCreate(&streamA);
        hipStreamCreate(&streamB);

        // A GRAD

        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridSize((x->mat_shape->x + BLOCK_SIZE - 1) / BLOCK_SIZE, (tensor->grad_shape->x + BLOCK_SIZE - 1) / BLOCK_SIZE);
        matmul_a_grad<<<gridSize, blockSize, 0, streamA>>>(gpu_tensor_grad, gpu_x, gpu_a_grad, tensor->grad_shape->x, tensor->grad_shape->y, x->mat_shape->x);
        checkCudaErr(hipPeekAtLastError());

        checkCudaErr(hipMemcpy(&a->grad[0], gpu_a_grad, a_size, hipMemcpyDeviceToHost));

        // X GRAD

        gridSize.x = (a->mat_shape->y + BLOCK_SIZE - 1) / BLOCK_SIZE;
        gridSize.y = (tensor->grad_shape->y + BLOCK_SIZE - 1) / BLOCK_SIZE;
        matmul_b_grad<<<gridSize, blockSize, 0, streamB>>>(gpu_tensor_grad, gpu_a, gpu_x_grad, tensor->grad_shape->y, tensor->grad_shape->x, a->mat_shape->y);
        checkCudaErr(hipPeekAtLastError());

        checkCudaErr(hipMemcpy(&x->grad[0], gpu_x_grad, x_size, hipMemcpyDeviceToHost));

        // B GRAD

        blockSize.x = BLOCK_SIZE_SUM;
        blockSize.y = 1;
        gridSize.x = (b->mat_shape->y + BLOCK_SIZE_SUM - 1) / BLOCK_SIZE_SUM;
        gridSize.y = 1;
        sum0<<<gridSize, blockSize>>>(gpu_b, gpu_b_grad, b->mat_shape->y, b->mat_shape->x);
        checkCudaErr(hipPeekAtLastError());

        checkCudaErr(hipMemcpy(&b->grad[0], gpu_b_grad, b_grad_size, hipMemcpyDeviceToHost));

        hipStreamDestroy(streamA);
        hipStreamDestroy(streamB);

        hipFree(gpu_tensor_grad);
        hipFree(gpu_a);
        hipFree(gpu_x);
        hipFree(gpu_b);
        hipFree(gpu_a_grad);
        hipFree(gpu_x_grad);
        hipFree(gpu_b_grad);

        return 0;
    }

}

