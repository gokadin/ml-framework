#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "tensor.h"

const int BLOCK_SIZE = 32;

__global__ void linear(float *a, float *x, float *b, float *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * x[i * k + col];
        }
        c[row * k + col] = sum + b[col];
    }
}

extern "C" {

    void linear(const TENSOR *a, const TENSOR *x, const TENSOR* b, TENSOR *target)
    {
        float* gpu_a;
        size_t size = a->shapeX * a->shapeY * sizeof(float);
        hipMalloc((void**)&gpu_a, size);
        hipMemcpy(gpu_a, &a->data[0], size, hipMemcpyHostToDevice);

        float* gpu_b;
        size_t size_b = b->shapeY * sizeof(float);
        hipMalloc((void**)&gpu_b, size_b);
        hipMemcpy(gpu_b, &b->data[0], size_b, hipMemcpyHostToDevice);

        float* gpu_x;
        size = x->shapeX * x->shapeY * sizeof(float);
        hipMalloc((void**)&gpu_x, size);
        hipMemcpy(gpu_x, &x->data[0], size, hipMemcpyHostToDevice);

        float* gpu_target;
        size = target->shapeX * target->shapeY * sizeof(float);
        hipMalloc(&gpu_target, size);

        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridSize((x->shapeY + BLOCK_SIZE - 1) / BLOCK_SIZE, (a->shapeX + BLOCK_SIZE - 1) / BLOCK_SIZE);
        linear<<<gridSize, blockSize>>>(gpu_a, gpu_x, gpu_b, gpu_target, a->shapeX, a->shapeY, x->shapeY);

        hipMemcpy(&target->data[0], gpu_target, size, hipMemcpyDeviceToHost);

        hipFree(gpu_a);
        hipFree(gpu_x);
        hipFree(gpu_b);
        hipFree(gpu_target);
    }

}

