#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cudautils.h"
#include "tensor.h"

const int BLOCK_SIZE = 1024;

__global__ void relu(float *a, float *target, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < size; i += stride) {
        if (a[i] > 0) {
            target[i] = a[i];
        } else {
            target[i] = 0;
        }
    }
}

__global__ void relu_grad(float *cg, float *ag, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < size; i += stride) {
        if (cg[i] > 0) {
            ag[i] = 1;
        } else {
            ag[i] = 0;
        }
    }
}

extern "C" {

    void gpu_relu_forward(TENSOR *target, TENSOR *a) {
        float* gpu_a;
        size_t a_size = a->mat_shape->size * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_a, a_size));
        checkCudaErr(hipMemcpy(gpu_a, &a->data[0], a_size, hipMemcpyHostToDevice));

        float* gpu_target;
        size_t target_size = target->mat_shape->size * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_target, target_size));

        dim3 blockSize = dim3(BLOCK_SIZE);
        dim3 gridSize = dim3((a->mat_shape->size + BLOCK_SIZE - 1) / BLOCK_SIZE);
        relu<<<gridSize, blockSize>>>(gpu_a, gpu_target, a->mat_shape->size);
        checkCudaKernelErr("relu", blockSize, gridSize);

        checkCudaErr(hipMemcpy(&target->data[0], gpu_target, target_size, hipMemcpyDeviceToHost));

        hipFree(gpu_a);
        hipFree(gpu_target);
    }

    void gpu_relu_backward(TENSOR *tensor, TENSOR *a) {
        float* gpu_tensor_grad;
        size_t tensor_grad_size = tensor->grad_shape->size * sizeof(float);
        checkCudaErr(hipMalloc((void**)&gpu_tensor_grad, tensor_grad_size));
        checkCudaErr(hipMemcpy(gpu_tensor_grad, &tensor->grad[0], tensor_grad_size, hipMemcpyHostToDevice));

        float* gpu_a_grad;
        size_t a_grad_size = a->grad_shape->size * sizeof(float);
        checkCudaErr(hipMalloc(&gpu_a_grad, a_grad_size));

        dim3 blockSize(BLOCK_SIZE);
        dim3 gridSize = dim3((a->grad_shape->size + BLOCK_SIZE - 1) / BLOCK_SIZE);
        relu_grad<<<gridSize, blockSize>>>(gpu_tensor_grad, gpu_a_grad, a->grad_shape->size);
        checkCudaKernelErr("relu_grad", blockSize, gridSize);

        checkCudaErr(hipMemcpy(&a->grad[0], gpu_a_grad, a_grad_size, hipMemcpyDeviceToHost));

        hipFree(gpu_tensor_grad);
        hipFree(gpu_a_grad);
    }

}

