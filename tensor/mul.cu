#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "tensor.h"

const int BLOCK_SIZE = 1024;

__global__ void mul(float *a, float *b, float *c, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
    {
        c[index] = a[index] * b[index];
    }
}

extern "C" {

    void mul(const TENSOR *a, const TENSOR* b, TENSOR *target) {
        float* gpu_a;
        size_t size = a->mat_shape->x * a->mat_shape->y * sizeof(float);
        hipMalloc((void**)&gpu_a, size);
        hipMemcpy(gpu_a, &a->data[0], size, hipMemcpyHostToDevice);

        float* gpu_b;
        size = b->mat_shape->x * b->mat_shape->y * sizeof(float);
        hipMalloc((void**)&gpu_b, size);
        hipMemcpy(gpu_b, &b->data[0], size, hipMemcpyHostToDevice);

        float* gpu_target;
        size = target->mat_shape->x * target->mat_shape->y * sizeof(float);
        hipMalloc(&gpu_target, size);

        dim3 blockSize(BLOCK_SIZE);
        dim3 gridSize((a->mat_shape->y * a->mat_shape->x + BLOCK_SIZE - 1) / BLOCK_SIZE);
        mul<<<gridSize, blockSize>>>(gpu_a, gpu_b, gpu_target, a->mat_shape->x * a->mat_shape->y);

        hipMemcpy(&target->data[0], gpu_target, size, hipMemcpyDeviceToHost);

        hipFree(gpu_a);
        hipFree(gpu_b);
        hipFree(gpu_target);
    }

}

