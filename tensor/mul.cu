#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "tensor.h"

const int BLOCK_SIZE = 1024;

__global__ void mul(float *a, float *b, float *c, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
    {
        c[index] = a[index] * b[index];
    }
}

extern "C" {

    void mul(const TENSOR *a, const TENSOR* b, TENSOR *target) {
        float* gpu_a;
        size_t size = a->shapeX * a->shapeY * sizeof(float);
        hipMalloc((void**)&gpu_a, size);
        hipMemcpy(gpu_a, &a->data[0], size, hipMemcpyHostToDevice);

        float* gpu_b;
        size = b->shapeX * b->shapeY * sizeof(float);
        hipMalloc((void**)&gpu_b, size);
        hipMemcpy(gpu_b, &b->data[0], size, hipMemcpyHostToDevice);

        float* gpu_target;
        size = target->shapeX * target->shapeY * sizeof(float);
        hipMalloc(&gpu_target, size);

        dim3 blockSize(BLOCK_SIZE);
        dim3 gridSize((a->shapeY * a->shapeX + BLOCK_SIZE - 1) / BLOCK_SIZE);
        mul<<<gridSize, blockSize>>>(gpu_a, gpu_b, gpu_target, a->shapeX * a->shapeY);

        hipMemcpy(&target->data[0], gpu_target, size, hipMemcpyDeviceToHost);

        hipFree(gpu_a);
        hipFree(gpu_b);
        hipFree(gpu_target);
    }

}

