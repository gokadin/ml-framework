#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "tensor.h"

const int BLOCK_SIZE = 32;

__global__ void matmul(float *a, float *b, float *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

__global__ void transpose(float* mat_in, float* mat_out, int rows, int cols)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows)
    {
        unsigned int pos = idy * cols + idx;
        unsigned int trans_pos = idx * rows + idy;
        mat_out[trans_pos] = mat_in[pos];
    }
}

extern "C" {

    void matmul(const TENSOR *a, const TENSOR* b, TENSOR *target)
    {
        float* gpu_a;
        size_t size = a->shapeX * a->shapeY * sizeof(float);
        hipMalloc((void**)&gpu_a, size);
        hipMemcpy(gpu_a, &a->data[0], size, hipMemcpyHostToDevice);

        float* gpu_b;
        size = b->shapeX * b->shapeY * sizeof(float);
        hipMalloc((void**)&gpu_b, size);
        hipMemcpy(gpu_b, &b->data[0], size, hipMemcpyHostToDevice);

        float* gpu_target;
        size = target->shapeX * target->shapeY * sizeof(float);
        hipMalloc(&gpu_target, size);

        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridSize((b->shapeY + BLOCK_SIZE - 1) / BLOCK_SIZE, (a->shapeX + BLOCK_SIZE - 1) / BLOCK_SIZE);
        matmul<<<gridSize, blockSize>>>(gpu_a, gpu_b, gpu_target, a->shapeX, a->shapeY, b->shapeY);

        hipMemcpy(&target->data[0], gpu_target, size, hipMemcpyDeviceToHost);

        hipFree(gpu_a);
        hipFree(gpu_b);
        hipFree(gpu_target);
    }

    void matmul_backward(const TENSOR *tensor, const TENSOR *a, TENSOR *b)
    {
        float* gpu_tensor_grad;
        size_t gpu_tensor_grad_size = tensor->shapeX * tensor->shapeY * sizeof(float);
        hipMalloc((void**)&gpu_tensor_grad, gpu_tensor_grad_size);
        hipMemcpy(gpu_tensor_grad, &tensor->grad[0], gpu_tensor_grad_size, hipMemcpyHostToDevice);

        float* gpu_tensor_grad_transpose;
        size_t gpu_tensor_grad_transpose_size = tensor->shapeX * tensor->shapeY * sizeof(float);
        hipMalloc((void**)&gpu_tensor_grad_transpose, gpu_tensor_grad_transpose_size);

        float* gpu_a;
        size_t a_size = a->shapeX * a->shapeY * sizeof(float);
        hipMalloc((void**)&gpu_a, a_size);
        hipMemcpy(gpu_a, &a->data[0], a_size, hipMemcpyHostToDevice);

        float* gpu_a_grad;
        hipMalloc(&gpu_a_grad, a_size);

        float* gpu_b;
        size_t b_size = b->shapeX * b->shapeY * sizeof(float);
        hipMalloc((void**)&gpu_b, b_size);
        hipMemcpy(gpu_b, &b->data[0], b_size, hipMemcpyHostToDevice);

        float* gpu_b_grad;
        hipMalloc(&gpu_b_grad, b_size);

        float* gpu_b_grad_transpose;
        hipMalloc((void**)&gpu_b_grad_transpose, b_size);

        float* gpu_b_transpose;
        hipMalloc((void**)&gpu_b_transpose, b_size);

        // B TRANSPOSE

        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridSize((b->shapeY + BLOCK_SIZE - 1) / BLOCK_SIZE, (b->shapeX + BLOCK_SIZE - 1) / BLOCK_SIZE);
        transpose<<<gridSize, blockSize>>>(gpu_b, gpu_b_transpose, b->shapeX, b->shapeY);

        // A GRAD

        blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE);
        gridSize = dim3((b->shapeX + BLOCK_SIZE - 1) / BLOCK_SIZE, (tensor->shapeX + BLOCK_SIZE - 1) / BLOCK_SIZE);
        matmul<<<gridSize, blockSize>>>(gpu_tensor_grad, gpu_b_transpose, gpu_a_grad, tensor->shapeX, tensor->shapeY, b->shapeX);

        hipMemcpy(&a->grad[0], gpu_a_grad, a_size, hipMemcpyDeviceToHost);

        // TENSOR GRAD TRANSPOSE

        blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE);
        gridSize = dim3((tensor->shapeY + BLOCK_SIZE - 1) / BLOCK_SIZE, (tensor->shapeX + BLOCK_SIZE - 1) / BLOCK_SIZE);
        transpose<<<gridSize, blockSize>>>(gpu_tensor_grad, gpu_tensor_grad_transpose, tensor->shapeX, tensor->shapeY);

        // B GRAD

        blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE);
        gridSize = dim3((a->shapeY + BLOCK_SIZE - 1) / BLOCK_SIZE, (tensor->shapeY + BLOCK_SIZE - 1) / BLOCK_SIZE);
        matmul<<<gridSize, blockSize>>>(gpu_tensor_grad_transpose, gpu_a, gpu_b_grad, tensor->shapeY, tensor->shapeX, a->shapeY);

        // B GRAD TRANSPOSE

        blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE);
        gridSize = dim3((b->shapeX + BLOCK_SIZE - 1) / BLOCK_SIZE, (b->shapeY + BLOCK_SIZE - 1) / BLOCK_SIZE);
        transpose<<<gridSize, blockSize>>>(gpu_b_grad, gpu_b_grad_transpose, b->shapeY, b->shapeX);

        hipMemcpy(&b->grad[0], gpu_b_grad_transpose, b_size, hipMemcpyDeviceToHost);

        hipFree(gpu_tensor_grad);
        hipFree(gpu_tensor_grad_transpose);
        hipFree(gpu_a);
        hipFree(gpu_b);
        hipFree(gpu_b_transpose);
        hipFree(gpu_b_grad_transpose);
        hipFree(gpu_a_grad);
        hipFree(gpu_b_grad);
    }

}

