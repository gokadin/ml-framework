#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "tensor.h"

const int BLOCK_SIZE = 32;

__global__ void matmul(float *a, float *b, float *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

__global__ void transpose(float* a, float* c, int rows, int cols)
{
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < cols && row < rows)
    {
        unsigned int pos = row * cols + col;
        unsigned int trans_pos = col * rows + row;
        c[trans_pos] = a[pos];
    }
}

__global__ void matmul_a_grad(float *cg, float *b, float *ag, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += cg[row * n + i] * b[col * n + i];
        }
        ag[row * k + col] = sum;
    }
}

__global__ void matmul_b_grad(float *cg, float *a, float *bg, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += cg[i * m + row] * a[i * k + col];
        }
        bg[col * m + row] = sum;
    }
}

extern "C" {

    void gpu_matmul_forward(const TENSOR *a, const TENSOR* b, TENSOR *target)
    {
        float* gpu_a;
        size_t size = a->mat_shape->x * a->mat_shape->y * sizeof(float);
        hipMalloc((void**)&gpu_a, size);
        hipMemcpy(gpu_a, &a->data[0], size, hipMemcpyHostToDevice);

        float* gpu_b;
        size = b->mat_shape->x * b->mat_shape->y * sizeof(float);
        hipMalloc((void**)&gpu_b, size);
        hipMemcpy(gpu_b, &b->data[0], size, hipMemcpyHostToDevice);

        float* gpu_target;
        size = target->mat_shape->x * target->mat_shape->y * sizeof(float);
        hipMalloc(&gpu_target, size);

        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridSize((b->mat_shape->y + BLOCK_SIZE - 1) / BLOCK_SIZE, (a->mat_shape->x + BLOCK_SIZE - 1) / BLOCK_SIZE);
        matmul<<<gridSize, blockSize>>>(gpu_a, gpu_b, gpu_target, a->mat_shape->x, a->mat_shape->y, b->mat_shape->y);

        hipMemcpy(&target->data[0], gpu_target, size, hipMemcpyDeviceToHost);

        hipFree(gpu_a);
        hipFree(gpu_b);
        hipFree(gpu_target);
    }

    void gpu_matmul_backward(const TENSOR *tensor, const TENSOR *a, TENSOR *b)
    {
        float* gpu_tensor_grad;
        size_t gpu_tensor_grad_size = tensor->grad_shape->x * tensor->grad_shape->y * sizeof(float);
        hipMalloc((void**)&gpu_tensor_grad, gpu_tensor_grad_size);
        hipMemcpy(gpu_tensor_grad, &tensor->grad[0], gpu_tensor_grad_size, hipMemcpyHostToDevice);

        float* gpu_a;
        size_t a_size = a->mat_shape->x * a->mat_shape->y * sizeof(float);
        hipMalloc((void**)&gpu_a, a_size);
        hipMemcpy(gpu_a, &a->data[0], a_size, hipMemcpyHostToDevice);

        float* gpu_a_grad;
        hipMalloc(&gpu_a_grad, a_size);

        float* gpu_b;
        size_t b_size = b->mat_shape->x * b->mat_shape->y * sizeof(float);
        hipMalloc((void**)&gpu_b, b_size);
        hipMemcpy(gpu_b, &b->data[0], b_size, hipMemcpyHostToDevice);

        float* gpu_b_grad;
        hipMalloc(&gpu_b_grad, b_size);

        hipStream_t streamA, streamB;
        hipStreamCreate(&streamA);
        hipStreamCreate(&streamB);

        // A GRAD

        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridSize((b->mat_shape->x + BLOCK_SIZE - 1) / BLOCK_SIZE, (tensor->grad_shape->x + BLOCK_SIZE - 1) / BLOCK_SIZE);
        matmul_a_grad<<<gridSize, blockSize, 0, streamA>>>(gpu_tensor_grad, gpu_b, gpu_a_grad, tensor->grad_shape->x, tensor->grad_shape->y, b->mat_shape->x);

        hipMemcpy(&a->grad[0], gpu_a_grad, a_size, hipMemcpyDeviceToHost);

        // B GRAD

        blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE);
        gridSize = dim3((a->mat_shape->y + BLOCK_SIZE - 1) / BLOCK_SIZE, (tensor->grad_shape->y + BLOCK_SIZE - 1) / BLOCK_SIZE);
        matmul_b_grad<<<gridSize, blockSize, 0, streamB>>>(gpu_tensor_grad, gpu_a, gpu_b_grad, tensor->grad_shape->y, tensor->grad_shape->x, a->mat_shape->y);

        hipMemcpy(&b->grad[0], gpu_b_grad, b_size, hipMemcpyDeviceToHost);

        hipStreamDestroy(streamA);
        hipStreamDestroy(streamB);

        hipFree(gpu_tensor_grad);
        hipFree(gpu_a);
        hipFree(gpu_b);
        hipFree(gpu_a_grad);
        hipFree(gpu_b_grad);
    }

}

