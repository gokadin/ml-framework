#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "tensor.h"

__global__ void cross_entropy(float *a, float* b, float *target, int size_a_x, int size_a_y)
{
    extern __shared__ float r[];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    r[index] = a[index];
    __syncthreads();

    for (int s = 0; s < size_a_y; s++) {
        int i = index / size_a_y;
        int j = i * size_a_y + s;
        target[i] += r[j];
        __syncthreads();
    }
}

extern "C" {

    void cross_entropy(TENSOR *a, TENSOR* b, TENSOR *target) {
        int size = a->shapeX * a->shapeY;
        int msize = size * sizeof(float);
        int msize_target = a->shapeX * sizeof(float);
        float* gpu_a;
        float* gpu_b;
        float* gpu_target;

        hipMalloc((void**)&gpu_a, msize);
        hipMemcpy(gpu_a, &a->data[0], msize, hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_b, msize);
        hipMemcpy(gpu_b, &b->data[0], msize, hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_target, msize_target);

        dim3 blockSize = dim3(size);
        dim3 gridSize = dim3((size + blockSize.x - 1) / blockSize.x);
        cross_entropy<<<gridSize, blockSize, msize>>>(gpu_a, gpu_b, gpu_target, a->shapeX, a->shapeY);

        hipMemcpy(&target->data[0], gpu_target, msize_target, hipMemcpyDeviceToHost);

        hipFree(gpu_a);
        hipFree(gpu_b);
        hipFree(gpu_target);
    }

}

